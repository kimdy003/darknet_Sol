#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include "avgpool_layer.h"
#include "hip/hip_runtime.h"
}

__global__ void forward_avgpool_layer_kernel(int n, int w, int h, int c, float *input, float *output)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(id >= n) return;

    int k = id % c;
    id /= c;
    int b = id;

    int i;
    int out_index = (k + c*b);
    output[out_index] = 0;
    for(i = 0; i < w*h; ++i){
        int in_index = i + h*w*(k + b*c);
        output[out_index] += input[in_index];
    }
    output[out_index] /= w*h;
}

__global__ void backward_avgpool_layer_kernel(int n, int w, int h, int c, float *in_delta, float *out_delta)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(id >= n) return;

    int k = id % c;
    id /= c;
    int b = id;

    int i;
    int out_index = (k + c*b);
    for(i = 0; i < w*h; ++i){
        int in_index = i + h*w*(k + b*c);
        in_delta[in_index] += out_delta[out_index] / (w*h);
    }
}

extern "C" void forward_avgpool_layer_gpu(avgpool_layer layer, network net)
{
    size_t n = layer.c*layer.batch;

    forward_avgpool_layer_kernel<<<cuda_gridsize(n), BLOCK>>>(n, layer.w, layer.h, layer.c, net.input_gpu, layer.output_gpu);
    check_error(hipPeekAtLastError());
}
#ifdef THREAD
extern "C" void forward_avgpool_layer_gpu_thread(netlayer* input, int id)
{
    network net = input->net;
    layer layer = input->layer;

    size_t n = layer.c*layer.batch;

    #ifdef STREAM
        //stream apply avgpool
        //fprintf(stderr, "[%d] index, avgpool id parameter : [%d] \n", net.index_n,  id);
        forward_avgpool_layer_kernel<<<cuda_gridsize(n), BLOCK, 0, usedstream(id)>>>(n, layer.w, layer.h, layer.c, net.input_gpu, layer.output_gpu);
        cuda_synchronize(id, __LINE__);
    #else
        forward_avgpool_layer_kernel<<<cuda_gridsize(n), BLOCK>>>(n, layer.w, layer.h, layer.c, net.input_gpu, layer.output_gpu);
    #endif
    check_error(hipPeekAtLastError());

}
#endif

extern "C" void backward_avgpool_layer_gpu(avgpool_layer layer, network net)
{
    size_t n = layer.c*layer.batch;

    backward_avgpool_layer_kernel<<<cuda_gridsize(n), BLOCK>>>(n, layer.w, layer.h, layer.c, net.delta_gpu, layer.delta_gpu);
    check_error(hipPeekAtLastError());
}

